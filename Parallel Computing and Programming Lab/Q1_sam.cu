// Q1. Write program in CUDA to add 2 vectors of length N using 
// A) Block Size as N
// B) N Threads

#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void add(int *a, int *b, int *c, int n) {
    // Perform the addition
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if(globalIdx < n){
        c[globalIdx] = a[globalIdx] + b[globalIdx];
    }
}

int main(void) {
    int n; // size of array
    int *h_a, *h_b, *h_c, *d_a, *d_b, *d_c; // device copies of variables a, b & c
    
    n=12;

    int size = n * sizeof(int);

    // HOST Memory allocated
    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);

    // Allocate space for device copies of h_a, h_b, h_c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Initializing the Array's
    for(int i=0 ; i<size; i++){
        h_a[i] = i;
        h_b[i] = 5*i;
    }

    // Copy inputs to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);


    // Launch add() kernel on GPU
    add<<< 1, n >>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(&h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print the result
    printf("Array: ");
    for(int i = 0 ; i<n ; i++){
        printf("%d ", h_c[i]);
    }

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
