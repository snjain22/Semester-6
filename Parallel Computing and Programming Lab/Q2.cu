#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10000  // Vector size (change as needed)
#define THREADS_PER_BLOCK 256  // Fixed number of threads per block

// CUDA Kernel for vector addition
__global__ void vectorAdd(int *A, int *B, int *C, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Global index

    if (idx < n) {  // Ensure within bounds
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int *h_A, *h_B, *h_C;  // Host vectors
    int *d_A, *d_B, *d_C;  // Device vectors
    int size = N * sizeof(int);

    // Allocate host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    // Initialize vectors with random values
    for (int i = 0; i < N; i++) {
        h_A[i] = rand() % 100;
        h_B[i] = rand() % 100;
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Calculate grid size (number of blocks)
    int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel with variable blocks and fixed 256 threads per block
    vectorAdd<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print some values for verification
    printf("Sample Results:\n");
    for (int i = 0; i < 5; i++) {
        printf("%d + %d = %d\n", h_A[i], h_B[i], h_C[i]);
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
