#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 10000  // Number of angles (change as needed)
#define THREADS_PER_BLOCK 256  // Fixed number of threads per block

// CUDA Kernel to compute sine of angles
__global__ void computeSine(float *angles, float *sine_values, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Compute global index

    if (idx < n) {  // Ensure within bounds
        sine_values[idx] = sinf(angles[idx]);  // Compute sine using CUDA's sinf function
    }
}

int main() {
    float *h_angles, *h_sine_values;  // Host arrays
    float *d_angles, *d_sine_values;  // Device arrays
    int size = N * sizeof(float);

    // Allocate host memory
    h_angles = (float*)malloc(size);
    h_sine_values = (float*)malloc(size);

    // Initialize angles in radians (e.g., 0 to 2π)
    for (int i = 0; i < N; i++) {
        h_angles[i] = ((float)i / N) * 2.0f * M_PI;  // Generating values between 0 and 2π
    }

    // Allocate device memory
    hipMalloc((void**)&d_angles, size);
    hipMalloc((void**)&d_sine_values, size);

    // Copy data from host to device
    hipMemcpy(d_angles, h_angles, size, hipMemcpyHostToDevice);

    // Calculate grid size (number of blocks)
    int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel with dynamically determined blocks and fixed 256 threads per block
    computeSine<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_angles, d_sine_values, N);

    // Copy result from device to host
    hipMemcpy(h_sine_values, d_sine_values, size, hipMemcpyDeviceToHost);

    // Print some sample results for verification
    printf("Angle (radians) | Sine Value\n");
    printf("-----------------------------\n");
    for (int i = 0; i < 5; i++) {
        printf("%.4f        | %.4f\n", h_angles[i], h_sine_values[i]);
    }

    // Free device memory
    hipFree(d_angles);
    hipFree(d_sine_values);

    // Free host memory
    free(h_angles);
    free(h_sine_values);

    return 0;
}
