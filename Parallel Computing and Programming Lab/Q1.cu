#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024  // Define the size of the vectors

// CUDA Kernel for vector addition
__global__ void vectorAdd(int *A, int *B, int *C, int n) {
    int idx = threadIdx.x;  // Get thread index within the block

    if (idx < n) {  // Ensure within bounds
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int *h_A, *h_B, *h_C;  // Host vectors
    int *d_A, *d_B, *d_C;  // Device vectors
    int size = N * sizeof(int);

    // Allocate host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    // Initialize vectors with random values
    for (int i = 0; i < N; i++) {
        h_A[i] = rand() % 100;
        h_B[i] = rand() % 100;
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel with one block and N threads
    vectorAdd<<<1, N>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print some values for verification
    printf("Sample Results:\n");
    for (int i = 0; i < 5; i++) {
        printf("%d + %d = %d\n", h_A[i], h_B[i], h_C[i]);
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
