#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10  // Number of elements in the array
#define THREADS_PER_BLOCK 256  // Maximum threads per block

// CUDA Kernel for Odd-Even Transposition Sort
__global__ void oddEvenSort(int *arr, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;  // Global thread index

    for (int phase = 0; phase < n; phase++) {
        if (phase % 2 == 0) {  // Even phase
            if (tid % 2 == 0 && tid + 1 < n) {
                if (arr[tid] > arr[tid + 1]) {
                    // Swap adjacent elements
                    int temp = arr[tid];
                    arr[tid] = arr[tid + 1];
                    arr[tid + 1] = temp;
                }
            }
        } else {  // Odd phase
            if (tid % 2 == 1 && tid + 1 < n) {
                if (arr[tid] > arr[tid + 1]) {
                    // Swap adjacent elements
                    int temp = arr[tid];
                    arr[tid] = arr[tid + 1];
                    arr[tid + 1] = temp;
                }
            }
        }
        __syncthreads();  // Synchronization after each phase
    }
}

int main() {
    int h_arr[N];  // Host array
    int *d_arr;  // Device array
    int size = N * sizeof(int);

    // Initialize array with random numbers
    printf("Original Array:\n");
    for (int i = 0; i < N; i++) {
        h_arr[i] = rand() % 100;  // Random numbers from 0-99
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    // Allocate device memory
    hipMalloc((void**)&d_arr, size);

    // Copy data from host to device
    hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice);

    // Calculate number of blocks
    int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel
    oddEvenSort<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_arr, N);

    // Copy sorted array back to host
    hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost);

    // Print sorted array
    printf("Sorted Array:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_arr);

    return 0;
}
