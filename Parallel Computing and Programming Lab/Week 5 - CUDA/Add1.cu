#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10000  // Vector size
#define THREADS_PER_BLOCK 256  // Fixed number of threads per block

// CUDA Kernel for the linear algebra operation: y = αx + y
__global__ void saxpy(float *x, float *y, float alpha, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Compute global index

    if (idx < n) {  // Ensure within bounds
        y[idx] = alpha * x[idx] + y[idx];  // Perform the operation
    }
}

int main() {
    float *h_x, *h_y;  // Host vectors
    float *d_x, *d_y;  // Device vectors
    float alpha = 2.5f;  // Scalar multiplier
    int size = N * sizeof(float);

    // Allocate host memory
    h_x = (float*)malloc(size);
    h_y = (float*)malloc(size);

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        h_x[i] = (float)(i % 100) * 0.1f;  // Some values for x
        h_y[i] = (float)(i % 50) * 0.2f;   // Some values for y
    }

    // Allocate device memory
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);

    // Copy data from host to device
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

    // Calculate grid size (number of blocks)
    int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel with variable blocks and fixed 256 threads per block
    saxpy<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_x, d_y, alpha, N);

    // Copy result from device to host
    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    // Print some sample results for verification
    printf("Sample Results (y = αx + y):\n");
    printf(" x[i]  | y[i] (updated)\n");
    printf("-------------------------\n");
    for (int i = 0; i < 5; i++) {
        printf("%.2f  | %.2f\n", h_x[i], h_y[i]);
    }

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);

    // Free host memory
    free(h_x);
    free(h_y);

    return 0;
}
