#include <stdio.h>
#include <hip/hip_runtime.h>

#define ROWS 5   // Number of rows in the matrix
#define COLS 8   // Number of columns in the matrix

// CUDA Kernel to perform selection sort on each row
__global__ void selectionSortRows(int *matrix, int rows, int cols) {
    int row = blockIdx.x;  // Each block handles one row

    if (row < rows) {  
        int *row_ptr = &matrix[row * cols];  // Get pointer to the row

        // Selection Sort for the row
        for (int i = 0; i < cols - 1; i++) {
            int minIdx = i;
            for (int j = i + 1; j < cols; j++) {
                if (row_ptr[j] < row_ptr[minIdx]) {
                    minIdx = j;
                }
            }
            // Swap row_ptr[i] and row_ptr[minIdx]
            if (minIdx != i) {
                int temp = row_ptr[i];
                row_ptr[i] = row_ptr[minIdx];
                row_ptr[minIdx] = temp;
            }
        }
    }
}

int main() {
    int h_matrix[ROWS][COLS];  // Host matrix
    int *d_matrix;  // Device matrix
    int size = ROWS * COLS * sizeof(int);

    // Initialize matrix with random values
    printf("Original Matrix:\n");
    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++) {
            h_matrix[i][j] = rand() % 100;  // Random numbers from 0-99
            printf("%2d ", h_matrix[i][j]);
        }
        printf("\n");
    }

    // Allocate device memory
    hipMalloc((void**)&d_matrix, size);

    // Copy matrix from host to device
    hipMemcpy(d_matrix, h_matrix, size, hipMemcpyHostToDevice);

    // Launch kernel with one block per row and COLS threads per block
    selectionSortRows<<<ROWS, 1>>>(d_matrix, ROWS, COLS);

    // Copy sorted matrix back to host
    hipMemcpy(h_matrix, d_matrix, size, hipMemcpyDeviceToHost);

    // Print sorted matrix
    printf("\nSorted Matrix (Each Row Sorted):\n");
    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++) {
            printf("%2d ", h_matrix[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_matrix);

    return 0;
}
